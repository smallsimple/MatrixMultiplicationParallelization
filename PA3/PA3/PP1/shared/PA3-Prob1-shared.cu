#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <sys/time.h>

#define N (2048)
// Use N=2048 for your testing
#define threshold (0.000000001)
double rtclock();
void compare();
double *A, *B, *RefOut;
#define bSize (64)
#define bSizeX (bSize)
#define bSizeY (bSize)
#define bNumX (N/bSizeX)
#define bNumY (N/bSizeY)


__global__ void p1_kernel(double*, double*, double*);

int main(){

    hipHostMalloc((void**)&A, N*N*sizeof(double), hipHostMallocDefault);
    hipHostMalloc((void**)&B, N*N*sizeof(double), hipHostMallocDefault);
    hipHostMalloc((void**)&RefOut, N*N*sizeof(double), hipHostMallocDefault);

    double clkbegin, clkend;
    double t;

    int i,j,k;
    printf("Matrix Size = %d\n",N);
    printf("dimGrid(%d,%d), dimBlock(%d,%d)\n",bNumX,bNumY,bSizeX,bSizeY);

    for(i=0;i<N;i++)
        for(j=0;j<N;j++)
        {
            A[i*N+j] = 1.0*(i+0.25*j)/(N+1);
            B[i*N+j] = 1.0*(i-0.5*j)/(N+1);
        }


    clkbegin = rtclock();
    //
    // Time the reference baseline version of code 
    // Its output is to be compared to the test version to verify correctness
    //
    for (i=0; i<N; i++)
        for (j=0; j<N; j++)
            for (k=i; k<N; k++)
                B[i*N+j] += A[i*N+k]*B[k*N+j];
    //
    // End of reference code
    //
    clkend = rtclock();
    t = clkend-clkbegin;
    if (B[(N/2)*N+(N/2)]*B[(N/2)*N+(N/2)] < -1000.0) 
        printf("To foil dead-code elimination by compiler: should never get here\n");
    printf("Base Sequential Symm-MatMult: %.1f GFLOPS; Time = %.3f sec; \n",
            1.0*N*N*(N+1)/t/1.0e9,t);

    for(i=0;i<N;i++)
        for(j=0;j<N;j++)
            RefOut[i*N+j] = B[i*N+j];

    // Initialize arrays

    for(i=0;i<N;i++)
        for(j=0;j<N;j++)
        {
            A[i*N+j] = 1.0*(i+0.25*j)/(N+1);
            B[i*N+j] = 1.0*(i-0.5*j)/(N+1);
        }

    double *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, N*N*sizeof(double));
    hipMalloc((void**)&d_B, N*N*sizeof(double));
    hipMalloc((void**)&d_C, N*N*sizeof(double));

    //
    // Test version of code; To be modified by you to improve performance
    //
    clkbegin = rtclock();

    // Send data to the GPU
    hipMemcpy(d_A, A, N*N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N*N*sizeof(double), hipMemcpyHostToDevice);
    
    // Set grid and block sizes
    dim3 dimGrid(bNumX,bNumX);
    dim3 dimBlock(bSizeX,bSizeY);

    // Call the kernel (located below, you must modify it too)
    p1_kernel<<< dimGrid, dimBlock >>>(d_A, d_B, d_C);

    // Retrieve results
    hipMemcpy(B, d_C, N*N*sizeof(double), hipMemcpyDeviceToHost);

    clkend = rtclock();

    t = clkend-clkbegin;
    if (B[(N/2)*N+(N/2)]*B[(N/2)*N+(N/2)] < -1000.0) 
        printf("To foil dead-code elimination by compiler: should never get here\n");
    printf ("%.1f GFLOPS; Time = %.3f sec; \n",
            1.0*N*N*(N+1)/t/1.0e9,t);

    //
    // Verify correctness by comparing result with reference version's
    //
    compare();
}

//
// Test version of the kernel; To be modified by you to improve performance
// 
__global__ void p1_kernel(double *A, double *B, double *C) {
    int bx=blockIdx.x; int by=blockIdx.y;
    int tx=threadIdx.x; int ty=threadIdx.y;
    
    int i = bSizeY*by+ty;
    int j = bSizeX*bx+tx;

    //int aBegin=N*bSizeY*by;
    //int bBegin=bSizeX*bx;
    
    __shared__ double As[bSizeX][bSizeY];
    __shared__ double Bs[bSizeX][bSizeY];

    int alnd=i*N+tx;
    int blnd=j+ty*N;

    double Bsub=B[i*N+j];

   for (int kt = 0; kt < N; kt+=bSizeX){

      As[ty][tx]=A[alnd];
      Bs[ty][tx]=B[blnd];
      __syncthreads();
        for (int k= (i>kt ? i : kt); k< kt+ bSizeX; ++k)
           Bsub+=As[ty][k-kt]*Bs[k-kt][tx];
      __syncthreads();
       alnd+=bSizeX;
       blnd+=bSizeY*N;
     }
    C[i*N+j]=Bsub;
}

double rtclock()
{
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday (&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d",stat);
    return(Tp.tv_sec + Tp.tv_usec*1.0e-6);
}


void compare()
{
    double maxdiff,this_diff;
    int numdiffs;
    int i,j;
    numdiffs = 0;
    maxdiff = 0;
    for (i=0;i<N;i++)
        for (j=0;j<N;j++)
        {
            this_diff = RefOut[i*N+j]-B[i*N+j];
            if (this_diff < 0) this_diff = -1.0*this_diff;
            if (this_diff>threshold)
            { numdiffs++;
               printf("i=%d j=%d  RefOut=%4.3f  B=%4.3f   diff=%.6f\n",i,j,RefOut[i*N+j],B[i*N+j],this_diff);
                if (this_diff > maxdiff) maxdiff=this_diff;
            }
        }
    if (numdiffs > 0)
        printf("%d Diffs found over threshold %f; Max Diff = %f\n",
                numdiffs,threshold,maxdiff);
    else
        printf("No differences found between base and test versions\n");
}

