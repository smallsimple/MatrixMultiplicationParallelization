#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <sys/time.h>

#define N (2048)
// Use N=2048 for your testing
//#define N (256) 
//#define N (256)
#define threshold (0.000000001)
double rtclock();
void compare();
double *A, *B, *RefOut;
#define bSizeX (256)
//#define bSizeY (1)
#define bNumX (N/bSizeX)
#define bNumY (N/2)

__global__ void p1_kernel(double*, double*, double*);

int main(){

    hipHostMalloc((void**)&A, N*N*sizeof(double), hipHostMallocDefault);
    hipHostMalloc((void**)&B, N*N*sizeof(double), hipHostMallocDefault);
    hipHostMalloc((void**)&RefOut, N*N*sizeof(double), hipHostMallocDefault);

    double clkbegin, clkend;
    double t;

    int i,j,k;
    printf("Matrix Size = %d\n",N);
    printf("dimGrid(%d,%d), dimBlock(%d,1)\n",bNumX,bNumY,bSizeX);

    for(i=0;i<N;i++)
        for(j=0;j<N;j++)
        {
            A[i*N+j] = 1.0*(i+0.25*j)/(N+1);
            B[i*N+j] = 1.0*(i-0.5*j)/(N+1);
        }


    clkbegin = rtclock();
    //
    // Time the reference baseline version of code 
    // Its output is to be compared to the test version to verify correctness
    //
    for (i=0; i<N; i++)
        for (j=0; j<N; j++)
            for (k=i; k<N; k++)
                B[i*N+j] += A[i*N+k]*B[k*N+j];
    //
    // End of reference code
    //
    clkend = rtclock();
    t = clkend-clkbegin;
    if (B[(N/2)*N+(N/2)]*B[(N/2)*N+(N/2)] < -1000.0) 
        printf("To foil dead-code elimination by compiler: should never get here\n");
    printf("Base Sequential Symm-MatMult: %.1f GFLOPS; Time = %.3f sec; \n",
            1.0*N*N*(N+1)/t/1.0e9,t);

    for(i=0;i<N;i++)
        for(j=0;j<N;j++)
            RefOut[i*N+j] = B[i*N+j];

    // Initialize arrays

    for(i=0;i<N;i++)
        for(j=0;j<N;j++)
        {
            A[i*N+j] = 1.0*(i+0.25*j)/(N+1);
            B[i*N+j] = 1.0*(i-0.5*j)/(N+1);
        }

    double *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, N*N*sizeof(double));
    hipMalloc((void**)&d_B, N*N*sizeof(double));
    hipMalloc((void**)&d_C, N*N*sizeof(double));

    //
    // Test version of code; To be modified by you to improve performance
    //
    clkbegin = rtclock();

    // Send data to the GPU
    hipMemcpy(d_A, A, N*N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N*N*sizeof(double), hipMemcpyHostToDevice);
    
    // Set grid and block sizes
    dim3 dimGrid(bNumX,bNumY);
    dim3 dimBlock(bSizeX,1);

    // Call the kernel (located below, you must modify it too)
    p1_kernel<<< dimGrid, dimBlock >>>(d_A, d_B, d_C);

    // Retrieve results
    hipMemcpy(B, d_C, N*N*sizeof(double), hipMemcpyDeviceToHost);

    clkend = rtclock();

    t = clkend-clkbegin;
    if (B[(N/2)*N+(N/2)]*B[(N/2)*N+(N/2)] < -1000.0) 
        printf("To foil dead-code elimination by compiler: should never get here\n");
    printf ("%.1f GFLOPS; Time = %.3f sec; \n",
            1.0*N*N*(N+1)/t/1.0e9,t);

    //
    // Verify correctness by comparing result with reference version's
    //
    compare();
}

//
// Test version of the kernel; To be modified by you to improve performance
// 
__global__ void p1_kernel(double *A, double *B, double *C) {
    int bx=blockIdx.x; int by=blockIdx.y;
    int tx=threadIdx.x;// int ty=threadIdx.y;

    int i = by*2;
    int j = bSizeX*bx+tx;
	
    __shared__ double As0[bSizeX],As1[bSizeX];
	
    double Bsub0=B[i*N+j];
	double Bsub1=B[(i+1)*N+j];
	Bsub0 += A[i*N+i]*B[i*N+j];
 	for(int kt=0; kt<N; kt+=bSizeX){
		As0[tx]=A[kt+tx+N*i];
		As1[tx]=A[kt+tx+N*(i+1)];
		__syncthreads();
		for(int k=(i>=kt ? i+1 : kt); k<kt+bSizeX; k++){
			double Br=B[k*N+j];
			Bsub0 += As0[k-kt]*Br;
			Bsub1 += As1[k-kt]*Br;
		}
		__syncthreads();
	} 
    C[i*N+j]=Bsub0;
    C[(i+1)*N+j]=Bsub1;
}

double rtclock()
{
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday (&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d",stat);
    return(Tp.tv_sec + Tp.tv_usec*1.0e-6);
}


void compare()
{
    double maxdiff,this_diff;
    int numdiffs;
    int i,j;
    numdiffs = 0;
    maxdiff = 0;
    for (i=0;i<N;i++)
        for (j=0;j<N;j++)
        {
            this_diff = RefOut[i*N+j]-B[i*N+j];
            if (this_diff < 0) this_diff = -1.0*this_diff;
            if (this_diff>threshold)
            { numdiffs++;
	//	printf("i=%d j=%d RefOut=%5.3lf  B=%5.3lf  diff=%5.3lf\n",i,j,RefOut[i*N+j],B[i*N+j]);
                if (this_diff > maxdiff) maxdiff=this_diff;
            }
        }
    if (numdiffs > 0)
        printf("%d Diffs found over threshold %f; Max Diff = %f\n",
                numdiffs,threshold,maxdiff);
    else
        printf("No differences found between base and test versions\n");
}

