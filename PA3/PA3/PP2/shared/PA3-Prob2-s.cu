#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <sys/time.h>

//#define N (2048)
// Use N=2048 for your testing
#define N (128) 
//#define N (4)
#define threshold (0.000000001)
double rtclock();
void compare();
double *A, *B, *C,*RefOut;
#define bSizeX (4)
#define bNumx (N/bSizeX)
#define bNumy (N)

__global__ void p1_kernel(double*, double*, double*);

int main(){

    hipHostMalloc((void**)&A, N*N*N*sizeof(double), hipHostMallocDefault);
    hipHostMalloc((void**)&B, N*N*N*sizeof(double), hipHostMallocDefault);
    hipHostMalloc((void**)&C, N*N*sizeof(double), hipHostMallocDefault);
    hipHostMalloc((void**)&RefOut, N*N*sizeof(double), hipHostMallocDefault);

    double clkbegin, clkend;
    double t;

    int i,j,k,l;
    printf("Matrix Size = %d\n",N);
    printf("dimGrid(%d,%d), dimBlock(%d,%d)\n",bNumx,bNumy,bSizeX,1);

    for(i=0;i<N;i++)
        for(j=0;j<N;j++) {
	  C[i*N+j] = 1.0*(i+0.25*j)/(N+1);
            for(k=0;k<N;k++)
            {
                A[i*N*N+j*N+k] = 1.0*(i+0.25*j-0.125*k)/(N+1);
                B[i*N*N+j*N+k] = 1.0*(i-0.5*j+0.125*k)/(N+1);
            }
        }

    clkbegin = rtclock();
    //
    // Time the reference baseline version of code 
    // Its output is to be compared to the test version to verify correctness
    //
    for (i=0; i<N; i++)
        for (j=0; j<N; j++)
            for (k=0; k<N; k++)
                for (l=k+1; l<N; l++)
                    C[k*N+l] += 0.5*(A[l*N*N+i*N+j]*B[k*N*N+i*N+j]+A[k*N*N+i*N+j]*B[l*N*N+i*N+j]);
    //
    // End of reference code
    //
    clkend = rtclock();
    t = clkend-clkbegin;
    if (C[(N/2)*N+(N/2)]*C[(N/2)*N+(N/2)] < -1000.0) 
        printf("To foil dead-code elimination by compiler: should never get here\n");
    printf("Base Sequential Symm-MatMult: %.1f GFLOPS; Time = %.3f sec; \n",
            5.0*N*N*N*(N+1)/2.0/t/1.0e9,t);

    for(i=0;i<N;i++)
        for(j=0;j<N;j++)
            RefOut[i*N+j] = C[i*N+j];

    // Initialize arrays

    for(i=0;i<N;i++)
        for(j=0;j<N;j++) {
	  C[i*N+j] = 1.0*(i+0.25*j)/(N+1);
            for(k=0;k<N;k++)
            {
                A[i*N*N+j*N+k] = 1.0*(i+0.25*j-0.125*k)/(N+1);
                B[i*N*N+j*N+k] = 1.0*(i-0.5*j+0.125*k)/(N+1);
            }
        }

    double *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, N*N*N*sizeof(double));
    hipMalloc((void**)&d_B, N*N*N*sizeof(double));
    hipMalloc((void**)&d_C, N*N*sizeof(double));

    //
    // Test version of code; To be modified by you to improve performance
    //
    clkbegin = rtclock();

    // Send data to the GPU
    hipMemcpy(d_A, A, N*N*N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N*N*N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, N*N*sizeof(double), hipMemcpyHostToDevice);
    
    // Set grid and block sizes
    dim3 dimGrid(bNumx,bNumy);
    dim3 dimBlock(bSizeX,1);

    // Call the kernel (located below, you must modify it too)
    p1_kernel<<< dimGrid, dimBlock >>>(d_A, d_B, d_C);

    // Retrieve results
    hipMemcpy(C, d_C, N*N*sizeof(double), hipMemcpyDeviceToHost);

    clkend = rtclock();

    t = clkend-clkbegin;
    if (C[(N/2)*N+(N/2)]*C[(N/2)*N+(N/2)] < -1000.0) 
        printf("To foil dead-code elimination by compiler: should never get here\n");
    printf ("%.1f GFLOPS; Time = %.3f sec; \n",
            5.0*N*N*N*(N+1)/2.0/t/1.0e9,t);

    //
    // Verify correctness by comparing result with reference version's
    //
    compare();
}

//
// Test version of the kernel; To be modified by you to improve performance
// 
__global__ void p1_kernel(double *A, double *B, double *C) {
  int bx=blockIdx.x; int by=blockIdx.y;
  int tx=threadIdx.x;
  
  int k = by;
  int l = bSizeX*bx+tx;
  int N2=N*N;

	__shared__ double As[bSizeX][N];
	__shared__ double Bs[bSizeX][N]; 

	double Csub = C[k*N+l];
	for (int is=0; is<N; is+= bSizeX){	
		for (int j=0; j<N; j++){
			As[tx][j] = A[k*N2+(is+tx)*N+j];
			Bs[tx][j] = B[k*N2+(is+tx)*N+j];
		}
		__syncthreads();
		if(k<l){ 
		        for (int i=is; i<is+bSizeX; i++)
				for (int j=0; j<N; j++)
        				Csub+=0.5*(As[i-is][j]*B[l*N2+i*N+j]+A[l*N2+i*N+j]*Bs[i-is][j]);
  		}
  	__syncthreads();
	}
	C[k*N+l]=Csub;
}

double rtclock()
{
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday (&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d",stat);
    return(Tp.tv_sec + Tp.tv_usec*1.0e-6);
}


void compare()
{
    double maxdiff,this_diff;
    int numdiffs;
    int i,j;
    numdiffs = 0;
    maxdiff = 0;
    for (i=0;i<N;i++)
        for (j=0;j<N;j++)
        {
            this_diff = RefOut[i*N+j]-C[i*N+j];
            if (this_diff < 0) this_diff = -1.0*this_diff;
            if (this_diff>threshold)
            { numdiffs++;
                if (this_diff > maxdiff) maxdiff=this_diff;
            }
        }
    if (numdiffs > 0)
        printf("%d Diffs found over threshold %f; Max Diff = %f\n",
                numdiffs,threshold,maxdiff);
    else
        printf("No differences found between base and test versions\n");
}

